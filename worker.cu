#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "tsp.h"

#define PI 3.14159265358979323846
#define TPB 1024
#define FLT_MAX (3.4e+38F)

__global__ void tourConstructionKernel(
    int* d_ant_tours,
    bool* d_ant_visited,
    float* d_choice_info,
    float* d_selection_probs,
    int num_ants,
    int num_cities,
    hiprandState* d_rand_state
) {
    int ant_idx = get_idx(num_ants);
    if (ant_idx >= num_ants) return;
    int i = ant_idx;

    int* ant_tour = &d_ant_tours[i * num_cities];
    bool* ant_visited = &d_ant_visited[i * num_cities];

    for (int j = 0; j < num_cities; ++j) ant_visited[j] = false;

    int current_city = i % num_cities;
    ant_tour[0] = current_city;
    ant_visited[current_city] = true;

    hiprandState local_state = d_rand_state[i];

    for (int step = 1; step < num_cities; ++step) {
        float sum_probs = 0.0f;
        float* selection_probs = &d_selection_probs[i * num_cities];

        for (int j = 0; j < num_cities; ++j) {
            if (!ant_visited[j]) {
                int idx = current_city * num_cities + j;
                selection_probs[j] = d_choice_info[idx];
                sum_probs += selection_probs[j];
            } else {
                selection_probs[j] = 0.0f;
            }
        }

        float r = hiprand_uniform(&local_state) * sum_probs;
        float accumulated_prob = 0.0f;
        int next_city = -1;

        for (int j = 0; j < num_cities; ++j) {
            if (selection_probs[j] > 0.0f && !ant_visited[j]) {
                accumulated_prob += selection_probs[j];
                if (accumulated_prob >= r) {
                    next_city = j;
                    break;
                }
            }
        }

        // if (next_city == -1) {
        //     printf("Error: No valid next city found for ant %d\n", i);
        // }

        // if (ant_visited[next_city]) {
        //     printf("Error! Choosing city which is already visited!\n");
        // }

        // if (next_city == current_city) {
        //     printf("Error! Choosing the same city again!\n");
        // }

        ant_tour[step] = next_city;
        ant_visited[next_city] = true;
        current_city = next_city;
    }

    d_rand_state[i] = local_state;
}

TspResult solveTSPWorker(
    const TspInput& tsp_input,
    unsigned int num_iter,
    float alpha,
    float beta,
    float evaporate,
    unsigned int seed
) {
    int num_cities = tsp_input.dimension;
    int num_ants = 128;
    size_t matrix_size = sizeof(float) * num_cities * num_cities;
    int* d_ant_tours;
    bool* d_ant_visited;
    hiprandState* d_rand_states;
    float* d_tour_lengths;
    float* d_choice_info;
    float* d_distances;
    float* d_selection_probs;
    float* d_pheromone;
    hipMalloc(&d_pheromone, matrix_size);
    hipMalloc(&d_ant_tours, sizeof(int) * num_ants * num_cities);
    hipMalloc(&d_ant_visited, sizeof(bool) * num_ants * num_cities);
    hipMalloc(&d_rand_states, sizeof(hiprandState) * num_ants);
    hipMalloc(&d_tour_lengths, sizeof(float) * num_ants);
    hipMalloc(&d_choice_info, matrix_size);
    hipMalloc(&d_distances, matrix_size);
    hipMalloc(&d_selection_probs, sizeof(float) * num_ants * num_cities);
    hipMemcpy(d_distances, tsp_input.distances, matrix_size, hipMemcpyHostToDevice);

    int num_blocks = (num_ants + TPB - 1) / TPB;
    assert(num_blocks * TPB >= num_ants);

    initialize_rand_states<<<num_blocks, TPB>>>(d_rand_states, num_ants, seed);
    hipDeviceSynchronize();

    // Initialize pheromones to 1.0
    float* h_initial_pheromone = new float[num_cities * num_cities];
    for (int i = 0; i < num_cities * num_cities; ++i) {
        h_initial_pheromone[i] = 1.0f;
    }
    hipMemcpy(d_pheromone, h_initial_pheromone, matrix_size, hipMemcpyHostToDevice);
    delete[] h_initial_pheromone;

#ifdef DEBUG
    float* h_choice_info = new float[num_cities * num_cities];
    for (int i=0; i < num_cities * num_cities; ++i) {
        h_choice_info[i] = PI;
    }
    hipMemcpy(d_choice_info, h_choice_info, matrix_size, hipMemcpyHostToDevice);
    printf("Before: d_choice_info[-1][-2] = %f\n", h_choice_info[num_cities * num_cities - 2]);
#endif
    computeChoiceInfoKernel<<<num_blocks, TPB>>>(d_choice_info, d_pheromone, d_distances, num_cities, alpha, beta);
    hipDeviceSynchronize();
#ifdef DEBUG
    hipMemcpy(h_choice_info, d_choice_info, matrix_size, hipMemcpyDeviceToHost);
    for (int i=0; i < num_cities * num_cities; ++i) {
        assert(h_choice_info[i] != PI);
    }
    printf("After: d_choice_info[-1][-2] = %f\n", h_choice_info[num_cities * num_cities - 2]);
#endif

    for (unsigned int iter = 0; iter < num_iter; ++iter) {
        computeChoiceInfoKernel<<<num_blocks, TPB>>>(d_choice_info, d_pheromone, d_distances, num_cities, alpha, beta);
        hipDeviceSynchronize();

        tourConstructionKernel<<<num_blocks, TPB>>>(
            d_ant_tours, d_ant_visited, d_choice_info, d_selection_probs, num_ants, num_cities, d_rand_states
        );
        hipDeviceSynchronize();

#ifdef DEBUG
        int* h_ant_tours = new int[num_ants * num_cities];
        hipMemcpy(h_ant_tours, d_ant_tours, sizeof(int) * num_ants * num_cities, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        verifyToursHost(h_ant_tours, num_ants, num_cities);
        delete[] h_ant_tours;
#endif

        evaporatePheromoneKernel<<<num_blocks, TPB>>>(d_pheromone, evaporate, num_cities);
        hipDeviceSynchronize();

        computeTourLengthsKernel<<<num_blocks, TPB>>>(
            d_ant_tours, d_distances, d_tour_lengths, num_ants, num_cities
        );
        hipDeviceSynchronize();

        depositPheromoneKernel<<<num_blocks, TPB>>>(
            d_pheromone, d_ant_tours, d_tour_lengths, num_ants, num_cities, 1.0 // TODO: what value Q here?
        );
        hipDeviceSynchronize();
    }

    float* h_tour_lengths = new float[num_ants];
    hipMemcpy(h_tour_lengths, d_tour_lengths, sizeof(float) * num_ants, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    float best_length = FLT_MAX;
    int best_idx = -1;
    for (int i = 0; i < num_ants; ++i) {
        if (h_tour_lengths[i] < best_length) {
            best_length = h_tour_lengths[i];
            best_idx = i;
        }
    }
    delete[] h_tour_lengths;

    if (best_idx == -1) {
        printf("Error: No valid ant found!\n");
        return {};
    }

    unsigned int* h_best_tour = new unsigned int[num_cities];
    hipMemcpy(h_best_tour, &d_ant_tours[best_idx * num_cities], sizeof(unsigned int) * num_cities, hipMemcpyDeviceToHost);

    TspResult result;
    result.dimension = num_cities;
    result.cost = best_length;
    result.tour = h_best_tour;

    hipFree(d_ant_tours);
    hipFree(d_pheromone);
    hipFree(d_ant_visited);
    hipFree(d_rand_states);
    hipFree(d_tour_lengths);
    hipFree(d_choice_info);
    hipFree(d_distances);
    hipFree(d_selection_probs);

    return result;
}